#include "hip/hip_runtime.h"
// Copyright (c) 2020 Michael Koesel and respective contributors
// SPDX-License-Identifier: MIT
// See accompanying LICENSE file for detailed information

#include "dogm/dogm_types.h"
#include "mapping/kernel/measurement_grid.h"

#include <hip/hip_runtime.h>
#include <>

#define PI 3.14159265358979323846f

__device__ float2 combine_masses(float2 prior, float2 meas)
{
    // Masses: mOcc, mFree
    float occ = prior.x;
    float free = prior.y;

    float meas_occ = meas.x;
    float meas_free = meas.y;

    float unknown_pred = 1.0f - occ - free;
    float meas_cell_unknown = 1.0f - meas_occ - meas_free;
    float K = free * meas_occ + occ * meas_free;

    float2 res;
    res.x = (occ * meas_cell_unknown + unknown_pred * meas_occ + occ * meas_occ) / (1.0f - K);
    res.y = (free * meas_cell_unknown + unknown_pred * meas_free + free * meas_free) / (1.0f - K);

    return res;
}

__device__ float pFree(int i, float p_min, float p_max, int max_range)
{
    return p_min + i * (p_max - p_min) / max_range;
}

__device__ float pOcc(int r, float zk, int index, float resolution)
{
    float occ_max = 0.95f;
    float delta = 0.6f / resolution;

    return occ_max * exp(-0.5f * (index - r) * (index - r) / (delta * delta));
}

__device__ float2 inverse_sensor_model(int i, float resolution, float zk, float r_max)
{
    // Masses: mOcc, mFree

    const float free = pFree(i, 0.15f, 1.0f, r_max);

    if (isfinite(zk))
    {
        const int r = static_cast<int>(zk / resolution);
        const float occ = pOcc(r, zk, i, resolution);

        if (i <= r)
        {
            return occ > free ? make_float2(occ, 0.0f) : make_float2(0.0f, 1.0f - free);
        }
        else
        {
            return occ > 0.5f ? make_float2(occ, 0.0f) : make_float2(0.0f, 0.0f);
        }
    }
    else
    {
        return make_float2(0.0f, 1.0f - free);
    }
}

__global__ void createPolarGridTextureKernel(hipSurfaceObject_t polar, const float* __restrict__ measurements,
                                             int width, int height, float resolution)
{
    const int theta = blockIdx.x * blockDim.x + threadIdx.x;
    const int range = blockIdx.y * blockDim.y + threadIdx.y;

    if (theta < width && range < height)
    {
        const float epsilon = 0.00001f;
        const float zk = measurements[theta];

        float2 masses = inverse_sensor_model(range, resolution, zk, height);
        masses.x = max(epsilon, min(1.0f - epsilon, masses.x));
        masses.y = max(epsilon, min(1.0f - epsilon, masses.y));

        surf2Dwrite(masses, polar, theta * sizeof(float2), range);
    }
}

__global__ void fusePolarGridTextureKernel(hipSurfaceObject_t polar, const float* __restrict__ measurements, int width,
                                           int height, float resolution)
{
    const int theta = blockIdx.x * blockDim.x + threadIdx.x;
    const int range = blockIdx.y * blockDim.y + threadIdx.y;

    if (theta < width && range < height)
    {
        const float epsilon = 0.00001f;
        const float zk = measurements[theta];

        float2 prior = surf2Dread<float2>(polar, theta * sizeof(float2), range);
        float2 masses = inverse_sensor_model(range, resolution, zk, height);
        masses.x = max(epsilon, min(1.0f - epsilon, masses.x));
        masses.y = max(epsilon, min(1.0f - epsilon, masses.y));

        float2 new_masses = combine_masses(prior, masses);
        // new_masses.x = max(epsilon, min(1.0f - epsilon, new_masses.x));
        // new_masses.y = max(epsilon, min(1.0f - epsilon, new_masses.y));

        surf2Dwrite(new_masses, polar, theta * sizeof(float2), range);
    }
}

__global__ void cartesianGridToMeasurementGridKernel(dogm::MeasurementCell* __restrict__ meas_grid,
                                                     hipSurfaceObject_t cart, int grid_size)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int index = (grid_size - y - 1) * grid_size + x;

    if (x < grid_size && y < grid_size)
    {
        float4 color = surf2Dread<float4>(cart, x * sizeof(float4), y);

        meas_grid[index].occ_mass = color.x;
        meas_grid[index].free_mass = color.y;

        meas_grid[index].likelihood = 1.0f;
        meas_grid[index].p_A = 1.0f;
    }
}

__global__ void gridArrayToMeasurementGridKernel(dogm::MeasurementCell* __restrict__ meas_grid,
                                                 const float2* __restrict__ grid, int grid_size)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int index = grid_size * y + x;

    if (x < grid_size && y < grid_size)
    {
        float2 masses = grid[index];

        meas_grid[index].occ_mass = masses.x;
        meas_grid[index].free_mass = masses.y;

        meas_grid[index].likelihood = 1.0f;
        meas_grid[index].p_A = 1.0f;
    }
}
