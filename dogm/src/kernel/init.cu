#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2019 Michael Kösel

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/
#include "dogm/common.h"
#include "dogm/cuda_utils.h"
#include "dogm/dogm_types.h"
#include "dogm/kernel/init.h"

#include <hip/hip_runtime.h>
#include <>

namespace dogm
{

__global__ void setupRandomStatesKernel(hiprandState* __restrict__ states, unsigned long long seed, int count)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x)
    {
        hiprand_init(seed, i, 0, &states[i]);
    }
}

__global__ void initParticlesKernel(Particle* __restrict__ particle_array, hiprandState* __restrict__ global_state,
                                    float velocity, int grid_size, int particle_count)
{
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    hiprandState local_state = global_state[thread_id];

    for (int i = thread_id; i < particle_count; i += stride)
    {
        float x = hiprand_uniform(&local_state, 0.0f, grid_size - 1);
        float y = hiprand_uniform(&local_state, 0.0f, grid_size - 1);
        float vel_x = hiprand_uniform(&local_state, -velocity, velocity);
        float vel_y = hiprand_uniform(&local_state, -velocity, velocity);

        particle_array[i].weight = 1.0f / particle_count;
        particle_array[i].state = glm::vec4(x, y, vel_x, vel_y);

        // printf("w: %f, x: %f, y: %f, vx: %f, vy: %f\n", particle_array[i].weight, particle_array[i].state[0],
        // particle_array[i].state[1], 	particle_array[i].state[2], particle_array[i].state[3]);
    }

    global_state[thread_id] = local_state;
}

__global__ void initBirthParticlesKernel(Particle* __restrict__ birth_particle_array,
                                         hiprandState* __restrict__ global_state, float velocity, int grid_size,
                                         int particle_count)
{
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // hiprandState local_state = global_state[thread_id];

    for (int i = thread_id; i < particle_count; i += stride)
    {
        // float x = hiprand_uniform(&local_state, 0.0f, grid_size - 1);
        // float y = hiprand_uniform(&local_state, 0.0f, grid_size - 1);
        // float vel_x = hiprand_normal(&local_state, 0.0f, velocity);
        // float vel_y = hiprand_normal(&local_state, 0.0f, velocity);

        birth_particle_array[i].weight = 0.0f;
        birth_particle_array[i].associated = false;
        birth_particle_array[i].state = glm::vec4(0.0f, 0.0f, 0.0f, 0.0f);
    }

    // global_state[thread_id] = local_state;
}

__global__ void initGridCellsKernel(GridCell* __restrict__ grid_cell_array,
                                    MeasurementCell* __restrict__ meas_cell_array, int grid_size, int cell_count)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < cell_count; i += blockDim.x * gridDim.x)
    {
        grid_cell_array[i].free_mass = 0.0f;
        grid_cell_array[i].occ_mass = 0.0f;
        grid_cell_array[i].start_idx = -1;
        grid_cell_array[i].end_idx = -1;

        meas_cell_array[i].occ_mass = 0.0f;
        meas_cell_array[i].free_mass = 0.0f;
        meas_cell_array[i].likelihood = 1.0f;
        meas_cell_array[i].p_A = 1.0f;
    }
}

__global__ void reinitGridParticleIndices(GridCell* __restrict__ grid_cell_array, int cell_count)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < cell_count; i += blockDim.x * gridDim.x)
    {
        grid_cell_array[i].start_idx = -1;
        grid_cell_array[i].end_idx = -1;
    }
}

} /* namespace dogm */
