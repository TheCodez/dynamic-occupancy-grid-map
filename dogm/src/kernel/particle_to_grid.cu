#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2019 Michael Kösel

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/
#include "dogm/cuda_utils.h"
#include "dogm/dogm_types.h"
#include "dogm/kernel/particle_to_grid.h"

#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

namespace dogm
{

__device__ bool is_first_particle(const Particle* __restrict__ particle_array, int i)
{
    return i == 0 || particle_array[i].grid_cell_idx != particle_array[i - 1].grid_cell_idx;
}

__device__ bool is_last_particle(const Particle* __restrict__ particle_array, int particle_count, int i)
{
    return i == particle_count - 1 || particle_array[i].grid_cell_idx != particle_array[i + 1].grid_cell_idx;
}

__global__ void particleToGridKernel(const Particle* __restrict__ particle_array,
                                     GridCell* __restrict__ grid_cell_array, float* __restrict__ weight_array,
                                     int particle_count)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < particle_count; i += blockDim.x * gridDim.x)
    {
        int j = particle_array[i].grid_cell_idx;

        if (is_first_particle(particle_array, i))
        {
            grid_cell_array[j].start_idx = i;
        }
        if (is_last_particle(particle_array, particle_count, i))
        {
            grid_cell_array[j].end_idx = i;
        }

        // printf("Cell: %d, Start idx: %d, End idx: %d\n", j, grid_cell_array[j].start_idx,
        // grid_cell_array[j].end_idx);
        weight_array[i] = particle_array[i].weight;
    }
}

} /* namespace dogm */
