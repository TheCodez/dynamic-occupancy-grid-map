#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2019 Michael Kösel

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/
#include "dogm/dogm_types.h"
#include "dogm/kernel/measurement_grid.h"

#include <hip/hip_runtime.h>
#include <>

#define PI 3.14159265358979323846f

namespace dogm
{

__device__ float2 combine_masses(float2 prior, float2 meas)
{
    // Masses: mOcc, mFree
    float occ = prior.x;
    float free = prior.y;

    float meas_occ = meas.x;
    float meas_free = meas.y;

    float unknown_pred = 1.0f - occ - free;
    float meas_cell_unknown = 1.0f - meas_occ - meas_free;
    float K = free * meas_occ + occ * meas_free;

    float2 res;
    res.x = (occ * meas_cell_unknown + unknown_pred * meas_occ + occ * meas_occ) / (1.0f - K);
    res.y = (free * meas_cell_unknown + unknown_pred * meas_free + free * meas_free) / (1.0f - K);

    return res;
}

__device__ float pFree(int i, float p_min, float p_max, int max_range)
{
    return p_min + i * (p_max - p_min) / max_range;
}

__device__ float pOcc(int r, float zk, int index)
{
    float alpha = 1.0f;
    float delta = 2.2f;

    // return (alpha / (delta * sqrt(2.0f * PI))) * exp(-0.5f * (index - r) * (index - r) / (delta * delta));
    return 0.8f * exp(-0.5f * (index - r) * (index - r) / (delta * delta));
}

__device__ float2 inverse_sensor_model(int i, float resolution, float zk, float r_max)
{
    // Masses: mOcc, mFree

    const float free = pFree(i, 0.1, 1.0f, r_max);

    if (isfinite(zk))
    {
        const int r = static_cast<int>(zk / resolution);
        const float occ = pOcc(r, zk, i);

        if (i <= r)
        {
            return occ > free ? make_float2(occ, 0.0f) : make_float2(0.0f, 1.0f - free);
        }
        else
        {
            return occ > 0.5f ? make_float2(occ, 0.0f) : make_float2(0.0f, 0.0f);
        }
    }
    else
    {
        return make_float2(0.0f, 1.0f - free);
    }
}

__global__ void createPolarGridTextureKernel(hipSurfaceObject_t polar, const float* __restrict__ measurements,
                                             int width, int height, float resolution)
{
    const int theta = blockIdx.x * blockDim.x + threadIdx.x;
    const int range = blockIdx.y * blockDim.y + threadIdx.y;

    if (theta < width && range < height)
    {
        const float epsilon = 0.00001f;
        const float zk = measurements[theta];

        float2 masses = inverse_sensor_model(range, resolution, zk, height);
        masses.x = max(epsilon, min(1.0f - epsilon, masses.x));
        masses.y = max(epsilon, min(1.0f - epsilon, masses.y));

        surf2Dwrite(masses, polar, theta * sizeof(float2), range);
    }
}

__global__ void createPolarGridTextureKernel2(hipSurfaceObject_t polar, MeasurementCell* __restrict__ polar_meas_grid,
                                              const float* __restrict__ measurements, int width, int height,
                                              float resolution)
{
    const int theta = blockIdx.x * blockDim.x + threadIdx.x;
    const int range = blockIdx.y * blockDim.y + threadIdx.y;

    if (theta < width && range < height)
    {
        const float epsilon = 0.00001f;
        const float zk = measurements[theta];

        float2 masses = inverse_sensor_model(range, resolution, zk, height);
        masses.x = max(epsilon, min(1.0f - epsilon, masses.x));
        masses.y = max(epsilon, min(1.0f - epsilon, masses.y));

        surf2Dwrite(masses, polar, theta * sizeof(float2), range);

        const int index = (height - range - 1) * width + theta;

        polar_meas_grid[index].occ_mass = masses.x;
        polar_meas_grid[index].free_mass = masses.y;
    }
}

__global__ void fusePolarGridTextureKernel(hipSurfaceObject_t polar, const float* __restrict__ measurements, int width,
                                           int height, float resolution)
{
    const int theta = blockIdx.x * blockDim.x + threadIdx.x;
    const int range = blockIdx.y * blockDim.y + threadIdx.y;

    if (theta < width && range < height)
    {
        const float epsilon = 0.00001f;
        const float zk = measurements[theta];

        float2 prior = surf2Dread<float2>(polar, theta * sizeof(float2), range);
        float2 masses = inverse_sensor_model(range, resolution, zk, height);
        masses.x = max(epsilon, min(1.0f - epsilon, masses.x));
        masses.y = max(epsilon, min(1.0f - epsilon, masses.y));

        float2 new_masses = combine_masses(prior, masses);
        // new_masses.x = max(epsilon, min(1.0f - epsilon, new_masses.x));
        // new_masses.y = max(epsilon, min(1.0f - epsilon, new_masses.y));

        surf2Dwrite(new_masses, polar, theta * sizeof(float2), range);
    }
}

__global__ void cartesianGridToMeasurementGridKernel(MeasurementCell* __restrict__ meas_grid, hipSurfaceObject_t cart,
                                                     int grid_size)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int index = (grid_size - y - 1) * grid_size + x;

    if (x < grid_size && y < grid_size)
    {
        float4 color = surf2Dread<float4>(cart, x * sizeof(float4), y);

        meas_grid[index].occ_mass = color.x;
        meas_grid[index].free_mass = color.y;

        meas_grid[index].likelihood = 1.0f;
        meas_grid[index].p_A = 1.0f;
    }
}

__global__ void gridArrayToMeasurementGridKernel(MeasurementCell* __restrict__ meas_grid,
                                                 const float2* __restrict__ grid, int grid_size)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int index = grid_size * y + x;

    if (x < grid_size && y < grid_size)
    {
        float2 masses = grid[index];

        meas_grid[index].occ_mass = masses.x;
        meas_grid[index].free_mass = masses.y;

        meas_grid[index].likelihood = 1.0f;
        meas_grid[index].p_A = 1.0f;
    }
}

} /* namespace dogm */
