#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2019 Michael Kösel

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/
#include "dogm/common.h"
#include "dogm/cuda_utils.h"
#include "dogm/dogm_types.h"
#include "dogm/kernel/init_new_particles.h"

#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>

namespace dogm
{

__device__ void set_cell_idx_A(Particle* __restrict__ birth_particle_array, int i, int grid_cell_idx)
{
    birth_particle_array[i].grid_cell_idx = grid_cell_idx;
    birth_particle_array[i].associated = true;
}

__device__ void set_cell_idx_UA(Particle* __restrict__ birth_particle_array, int i, int grid_cell_idx)
{
    birth_particle_array[i].grid_cell_idx = grid_cell_idx;
    birth_particle_array[i].associated = false;
}

__device__ int calc_start_idx(const float* __restrict__ particle_orders_array_accum, int index)
{
    if (index == 0)
    {
        return 0;
    }

    return static_cast<int>(particle_orders_array_accum[index - 1]);
}

__device__ int calc_end_idx(const float* __restrict__ particle_orders_array_accum, int index)
{
    return static_cast<int>(particle_orders_array_accum[index]) - 1;
}

__device__ int calc_num_assoc(int num_new_particles, float p_A)
{
    return static_cast<int>(roundf(num_new_particles * p_A));
}

__device__ float calc_weight_assoc(int nu_A, float p_A, float born_mass)
{
    return nu_A > 0 ? (p_A * born_mass) / nu_A : 0.0;
}

__device__ float calc_weight_unassoc(int nu_UA, float p_A, float born_mass)
{
    return nu_UA > 0 ? ((1.0 - p_A) * born_mass) / nu_UA : 0.0;
}

__device__ void store_weights(float w_A, float w_UA, GridCell* __restrict__ grid_cell_array, int j)
{
    grid_cell_array[j].w_A = w_A;
    grid_cell_array[j].w_UA = w_UA;
}

void normalize_particle_orders(float* particle_orders_array_accum, int particle_orders_count, int v_B)
{
    thrust::device_ptr<float> particle_orders_accum(particle_orders_array_accum);

    float max = 1.0f;
    hipMemcpy(&max, &particle_orders_array_accum[particle_orders_count - 1], sizeof(float), hipMemcpyDeviceToHost);
    thrust::transform(particle_orders_accum, particle_orders_accum + particle_orders_count, particle_orders_accum,
                      GPU_LAMBDA(float x) { return x * (v_B / max); });
}

__global__ void initNewParticlesKernel1(GridCell* __restrict__ grid_cell_array,
                                        const MeasurementCell* __restrict__ meas_cell_array,
                                        const float* __restrict__ weight_array,
                                        const float* __restrict__ born_masses_array,
                                        Particle* __restrict__ birth_particle_array,
                                        const float* __restrict__ particle_orders_array_accum, int cell_count)
{
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < cell_count; j += blockDim.x * gridDim.x)
    {
        int start_idx = calc_start_idx(particle_orders_array_accum, j);
        int end_idx = calc_end_idx(particle_orders_array_accum, j);

        // printf("Start idx: %d, End idx: %d\n", start_idx, end_idx);

        int num_new_particles = start_idx <= end_idx ? end_idx - start_idx + 1 : 0;
        float p_A = meas_cell_array[j].p_A;
        int nu_A = calc_num_assoc(num_new_particles, p_A);
        int nu_UA = num_new_particles - nu_A;
        float w_A = calc_weight_assoc(nu_A, p_A, born_masses_array[j]);
        float w_UA = calc_weight_unassoc(nu_UA, p_A, born_masses_array[j]);
        store_weights(w_A, w_UA, grid_cell_array, j);

        // printf("w_A: %f, w_UA: %f\n", w_A, w_UA);

        for (int i = start_idx; i < start_idx + nu_A + 1; i++)
        {
            set_cell_idx_A(birth_particle_array, i, j);
        }

        for (int i = start_idx + nu_A + 1; i < end_idx + 1; i++)
        {
            set_cell_idx_UA(birth_particle_array, i, j);
        }
    }
}

__global__ void initNewParticlesKernel2(Particle* __restrict__ birth_particle_array,
                                        const GridCell* __restrict__ grid_cell_array,
                                        hiprandState* __restrict__ global_state, float velocity, int grid_size,
                                        int particle_count)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < particle_count; i += blockDim.x * gridDim.x)
    {
        hiprandState local_state = global_state[i];

        int cell_idx = birth_particle_array[i].grid_cell_idx;
        const GridCell& grid_cell = grid_cell_array[cell_idx];

        float x = cell_idx % grid_size;
        float y = cell_idx / grid_size;
        float vel_x = hiprand_normal(&local_state, 0.0f, velocity);
        float vel_y = hiprand_normal(&local_state, 0.0f, velocity);

        bool associated = birth_particle_array[i].associated;
        // TODO: Use correct distribution
        if (associated)
        {
            birth_particle_array[i].weight = grid_cell.w_A;
            birth_particle_array[i].state = glm::vec4(x, y, vel_x, vel_y);
        }
        else
        {
            birth_particle_array[i].weight = grid_cell.w_UA;
            birth_particle_array[i].state = glm::vec4(x, y, vel_x, vel_y);
        }

        global_state[i] = local_state;
    }
}

__global__ void copyBirthWeightKernel(const Particle* __restrict__ birth_particle_array,
                                      float* __restrict__ birth_weight_array, int particle_count)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < particle_count; i += blockDim.x * gridDim.x)
    {
        birth_weight_array[i] = birth_particle_array[i].weight;
    }
}

} /* namespace dogm */
