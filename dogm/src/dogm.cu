#include "hip/hip_runtime.h"
// Copyright (c) 2020 Michael Koesel and respective contributors
// SPDX-License-Identifier: MIT
// See accompanying LICENSE file for detailed information

#include "dogm/common.h"
#include "dogm/cuda_utils.h"
#include "dogm/dogm.h"
#include "dogm/dogm_types.h"

#include "dogm/kernel/ego_motion_compensation.h"
#include "dogm/kernel/init.h"
#include "dogm/kernel/init_new_particles.h"
#include "dogm/kernel/mass_update.h"
#include "dogm/kernel/particle_to_grid.h"
#include "dogm/kernel/predict.h"
#include "dogm/kernel/resampling.h"
#include "dogm/kernel/statistical_moments.h"
#include "dogm/kernel/update_persistent_particles.h"

#include <thrust/sort.h>
#include <thrust/transform.h>

#include <hip/hip_runtime.h>

#include <vector>

namespace dogm
{

constexpr int BLOCK_SIZE = 256;

DOGM::DOGM(const Params& params)
    : params(params), grid_size(static_cast<int>(params.size / params.resolution)),
      particle_count(params.particle_count), grid_cell_count(grid_size * grid_size),
      new_born_particle_count(params.new_born_particle_count), block_dim(BLOCK_SIZE), first_pose_received(false),
      first_measurement_received(false), position_x(0.0f), position_y(0.0f)
{
    int device;
    CHECK_ERROR(hipGetDevice(&device));

    hipDeviceProp_t device_prop;
    CHECK_ERROR(hipGetDeviceProperties(&device_prop, device));

    int blocks_per_sm = device_prop.maxThreadsPerMultiProcessor / block_dim.x;
    dim3 dim(device_prop.multiProcessorCount * blocks_per_sm);
    particles_grid = birth_particles_grid = grid_map_grid = dim;

    particle_array.init(particle_count, true);
    particle_array_next.init(particle_count, true);
    birth_particle_array.init(new_born_particle_count, true);

    CHECK_ERROR(hipMalloc(&grid_cell_array, grid_cell_count * sizeof(GridCell)));
    CHECK_ERROR(hipMalloc(&meas_cell_array, grid_cell_count * sizeof(MeasurementCell)));

    CHECK_ERROR(hipMalloc(&weight_array, particle_count * sizeof(float)));
    CHECK_ERROR(hipMalloc(&birth_weight_array, new_born_particle_count * sizeof(float)));
    CHECK_ERROR(hipMalloc(&born_masses_array, grid_cell_count * sizeof(float)));

    CHECK_ERROR(hipMalloc(&vel_x_array, particle_count * sizeof(float)));
    CHECK_ERROR(hipMalloc(&vel_y_array, particle_count * sizeof(float)));
    CHECK_ERROR(hipMalloc(&vel_x_squared_array, particle_count * sizeof(float)));
    CHECK_ERROR(hipMalloc(&vel_y_squared_array, particle_count * sizeof(float)));
    CHECK_ERROR(hipMalloc(&vel_xy_array, particle_count * sizeof(float)));

    CHECK_ERROR(hipMalloc(&rand_array, particle_count * sizeof(float)));

    CHECK_ERROR(hipMalloc(&rng_states, particles_grid.x * block_dim.x * sizeof(hiprandState)));

    initialize();
}

DOGM::~DOGM()
{
    particle_array.free();
    particle_array_next.free();
    birth_particle_array.free();

    CHECK_ERROR(hipFree(grid_cell_array));
    CHECK_ERROR(hipFree(meas_cell_array));

    CHECK_ERROR(hipFree(weight_array));
    CHECK_ERROR(hipFree(birth_weight_array));
    CHECK_ERROR(hipFree(born_masses_array));

    CHECK_ERROR(hipFree(vel_x_array));
    CHECK_ERROR(hipFree(vel_y_array));
    CHECK_ERROR(hipFree(vel_x_squared_array));
    CHECK_ERROR(hipFree(vel_y_squared_array));
    CHECK_ERROR(hipFree(vel_xy_array));

    CHECK_ERROR(hipFree(rng_states));
}

void DOGM::initialize()
{
    hipStream_t particles_stream, grid_stream;
    CHECK_ERROR(hipStreamCreate(&particles_stream));
    CHECK_ERROR(hipStreamCreate(&grid_stream));

    setupRandomStatesKernel<<<particles_grid, block_dim>>>(rng_states, 123456, particles_grid.x * block_dim.x);

    CHECK_ERROR(hipGetLastError());
    CHECK_ERROR(hipDeviceSynchronize());

    initGridCellsKernel<<<grid_map_grid, block_dim, 0, grid_stream>>>(grid_cell_array, meas_cell_array, grid_size,
                                                                      grid_cell_count);

    CHECK_ERROR(hipGetLastError());

    CHECK_ERROR(hipStreamDestroy(particles_stream));
    CHECK_ERROR(hipStreamDestroy(grid_stream));
}

void DOGM::updateGrid(MeasurementCell* measurement_grid, float new_x, float new_y, float new_yaw, float dt, bool device)
{
    updateMeasurementGrid(measurement_grid, device);
    updatePose(new_x, new_y, new_yaw);

    particlePrediction(dt);
    particleAssignment();
    gridCellOccupancyUpdate();
    updatePersistentParticles();
    initializeNewParticles();
    statisticalMoments();
    resampling();

    particle_array = particle_array_next;

    CHECK_ERROR(hipDeviceSynchronize());
}

std::vector<GridCell> DOGM::getGridCells() const
{
    std::vector<GridCell> grid_cells(static_cast<std::vector<GridCell>::size_type>(grid_cell_count));

    CHECK_ERROR(
        hipMemcpy(grid_cells.data(), grid_cell_array, grid_cell_count * sizeof(GridCell), hipMemcpyDeviceToHost));

    return grid_cells;
}

std::vector<MeasurementCell> DOGM::getMeasurementCells() const
{
    std::vector<MeasurementCell> meas_cells(static_cast<std::vector<GridCell>::size_type>(grid_cell_count));

    CHECK_ERROR(hipMemcpy(meas_cells.data(), meas_cell_array, grid_cell_count * sizeof(MeasurementCell),
                           hipMemcpyDeviceToHost));

    return meas_cells;
}

ParticlesSoA DOGM::getParticles() const
{
    ParticlesSoA particles(particle_count, false);
    particles.copy(particle_array, hipMemcpyDeviceToHost);

    return particles;
}

void DOGM::updatePose(float new_x, float new_y, float new_yaw)
{
    if (!first_pose_received)
    {
        position_x = new_x;
        position_y = new_y;
        yaw = new_yaw;
        first_pose_received = true;
    }
    else
    {
        const float x_diff = new_x - position_x;
        const float y_diff = new_y - position_y;

        if (fabsf(x_diff) > params.resolution || fabsf(y_diff) > params.resolution)
        {
            const int x_move = -static_cast<int>(x_diff / params.resolution);
            const int y_move = -static_cast<int>(y_diff / params.resolution);

            GridCell* old_grid_cell_array;
            CHECK_ERROR(hipMalloc(&old_grid_cell_array, grid_cell_count * sizeof(GridCell)));

            CHECK_ERROR(hipMemcpy(old_grid_cell_array, grid_cell_array, grid_cell_count * sizeof(GridCell),
                                   hipMemcpyDeviceToDevice));
            CHECK_ERROR(hipMemset(grid_cell_array, 0, grid_cell_count * sizeof(GridCell)));

            dim3 dim_block(32, 32);
            dim3 grid_dim(divUp(grid_size, dim_block.x), divUp(grid_size, dim_block.y));

            moveParticlesKernel<<<particles_grid, block_dim>>>(particle_array, x_move, y_move, particle_count);
            CHECK_ERROR(hipGetLastError());

            moveMapKernel<<<grid_dim, dim_block>>>(grid_cell_array, old_grid_cell_array, x_move, y_move, grid_size);
            CHECK_ERROR(hipGetLastError());

            CHECK_ERROR(hipFree(old_grid_cell_array));

            position_x = new_x;
            position_y = new_y;
            yaw = new_yaw;
        }
    }
}

void DOGM::updateMeasurementGrid(MeasurementCell* measurement_grid, bool device)
{
    hipMemcpyKind kind = device ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
    CHECK_ERROR(hipMemcpy(meas_cell_array, measurement_grid, grid_cell_count * sizeof(MeasurementCell), kind));

    if (!first_measurement_received)
    {
        initializeParticles();
        first_measurement_received = true;
    }
}

void DOGM::initializeParticles()
{
    copyMassesKernel<<<grid_map_grid, block_dim>>>(meas_cell_array, born_masses_array, grid_cell_count);

    CHECK_ERROR(hipGetLastError());
    CHECK_ERROR(hipDeviceSynchronize());

    thrust::device_vector<float> particle_orders_accum(grid_cell_count);
    accumulate(born_masses_array, particle_orders_accum);
    float* particle_orders_array_accum = thrust::raw_pointer_cast(particle_orders_accum.data());

    float new_weight = 1.0f / particle_count;

    normalize_particle_orders(particle_orders_array_accum, grid_cell_count, particle_count);

    initParticlesKernel1<<<grid_map_grid, block_dim>>>(grid_cell_array, meas_cell_array, particle_array,
                                                       particle_orders_array_accum, grid_cell_count);

    CHECK_ERROR(hipGetLastError());

    initParticlesKernel2<<<particles_grid, block_dim>>>(
        particle_array, grid_cell_array, rng_states, params.init_max_velocity, grid_size, new_weight, particle_count);

    CHECK_ERROR(hipGetLastError());
}

void DOGM::particlePrediction(float dt)
{
    // clang-format off
    glm::mat4x4 transition_matrix(1, 0, dt, 0,
                                  0, 1, 0, dt,
                                  0, 0, 1, 0,
                                  0, 0, 0, 1);
    // clang-format on

    // FIXME: glm uses column major, we need row major
    transition_matrix = glm::transpose(transition_matrix);

    predictKernel<<<particles_grid, block_dim>>>(
        particle_array, rng_states, params.stddev_velocity, grid_size, params.persistence_prob, transition_matrix,
        params.stddev_process_noise_position, params.stddev_process_noise_velocity, particle_count);

    CHECK_ERROR(hipGetLastError());
}

void DOGM::particleAssignment()
{
    reinitGridParticleIndices<<<grid_map_grid, block_dim>>>(grid_cell_array, grid_cell_count);

    CHECK_ERROR(hipGetLastError());
    // CHECK_ERROR(hipDeviceSynchronize());

    // sort particles
    thrust::device_ptr<int> grid_index_ptr(particle_array.grid_cell_idx);
    thrust::device_ptr<float> weight_ptr(particle_array.weight);
    thrust::device_ptr<bool> associated_ptr(particle_array.associated);
    thrust::device_ptr<glm::vec4> state_ptr(particle_array.state);

    auto it = thrust::make_zip_iterator(thrust::make_tuple(weight_ptr, associated_ptr, state_ptr));
    thrust::sort_by_key(grid_index_ptr, grid_index_ptr + particle_count, it);

    particleToGridKernel<<<particles_grid, block_dim>>>(particle_array, grid_cell_array, weight_array, particle_count);

    CHECK_ERROR(hipGetLastError());
}

void DOGM::gridCellOccupancyUpdate()
{
    // CHECK_ERROR(hipDeviceSynchronize());

    thrust::device_vector<float> weights_accum(particle_count);
    accumulate(weight_array, weights_accum);
    float* weight_array_accum = thrust::raw_pointer_cast(weights_accum.data());

    gridCellPredictionUpdateKernel<<<grid_map_grid, block_dim>>>(grid_cell_array, particle_array, weight_array,
                                                                 weight_array_accum, meas_cell_array, born_masses_array,
                                                                 params.birth_prob, grid_cell_count);

    CHECK_ERROR(hipGetLastError());
}

void DOGM::updatePersistentParticles()
{
    updatePersistentParticlesKernel1<<<particles_grid, block_dim>>>(particle_array, meas_cell_array, weight_array,
                                                                    particle_count);

    CHECK_ERROR(hipGetLastError());
    // CHECK_ERROR(hipDeviceSynchronize());

    thrust::device_vector<float> weights_accum(particle_count);
    accumulate(weight_array, weights_accum);
    float* weight_array_accum = thrust::raw_pointer_cast(weights_accum.data());

    updatePersistentParticlesKernel2<<<divUp(grid_cell_count, BLOCK_SIZE), BLOCK_SIZE>>>(
        grid_cell_array, weight_array_accum, grid_cell_count);

    CHECK_ERROR(hipGetLastError());

    updatePersistentParticlesKernel3<<<particles_grid, block_dim>>>(particle_array, meas_cell_array, grid_cell_array,
                                                                    weight_array, particle_count);

    CHECK_ERROR(hipGetLastError());
}

void DOGM::initializeNewParticles()
{
    initBirthParticlesKernel<<<birth_particles_grid, block_dim>>>(
        birth_particle_array, rng_states, params.stddev_velocity, grid_size, new_born_particle_count);

    CHECK_ERROR(hipGetLastError());
    // CHECK_ERROR(hipDeviceSynchronize());

    thrust::device_vector<float> particle_orders_accum(grid_cell_count);
    accumulate(born_masses_array, particle_orders_accum);
    float* particle_orders_array_accum = thrust::raw_pointer_cast(particle_orders_accum.data());

    normalize_particle_orders(particle_orders_array_accum, grid_cell_count, new_born_particle_count);

    initNewParticlesKernel1<<<grid_map_grid, block_dim>>>(grid_cell_array, meas_cell_array, weight_array,
                                                          born_masses_array, birth_particle_array,
                                                          particle_orders_array_accum, grid_cell_count);

    CHECK_ERROR(hipGetLastError());

    initNewParticlesKernel2<<<birth_particles_grid, block_dim>>>(birth_particle_array, grid_cell_array, rng_states,
                                                                 params.stddev_velocity, params.init_max_velocity,
                                                                 grid_size, new_born_particle_count);

    CHECK_ERROR(hipGetLastError());
}

void DOGM::statisticalMoments()
{
    statisticalMomentsKernel1<<<particles_grid, block_dim>>>(particle_array, weight_array, vel_x_array, vel_y_array,
                                                             vel_x_squared_array, vel_y_squared_array, vel_xy_array,
                                                             particle_count);

    CHECK_ERROR(hipGetLastError());
    // CHECK_ERROR(hipDeviceSynchronize());

    thrust::device_vector<float> vel_x_accum(particle_count);
    accumulate(vel_x_array, vel_x_accum);
    float* vel_x_array_accum = thrust::raw_pointer_cast(vel_x_accum.data());

    thrust::device_vector<float> vel_y_accum(particle_count);
    accumulate(vel_y_array, vel_y_accum);
    float* vel_y_array_accum = thrust::raw_pointer_cast(vel_y_accum.data());

    thrust::device_vector<float> vel_x_squared_accum(particle_count);
    accumulate(vel_x_squared_array, vel_x_squared_accum);
    float* vel_x_squared_array_accum = thrust::raw_pointer_cast(vel_x_squared_accum.data());

    thrust::device_vector<float> vel_y_squared_accum(particle_count);
    accumulate(vel_y_squared_array, vel_y_squared_accum);
    float* vel_y_squared_array_accum = thrust::raw_pointer_cast(vel_y_squared_accum.data());

    thrust::device_vector<float> vel_xy_accum(particle_count);
    accumulate(vel_xy_array, vel_xy_accum);
    float* vel_xy_array_accum = thrust::raw_pointer_cast(vel_xy_accum.data());

    statisticalMomentsKernel2<<<grid_map_grid, block_dim>>>(grid_cell_array, vel_x_array_accum, vel_y_array_accum,
                                                            vel_x_squared_array_accum, vel_y_squared_array_accum,
                                                            vel_xy_array_accum, grid_cell_count);

    CHECK_ERROR(hipGetLastError());
}

void DOGM::resampling()
{
    // CHECK_ERROR(hipDeviceSynchronize());

    thrust::device_ptr<float> persistent_weights(weight_array);
    thrust::device_ptr<float> new_born_weights(birth_particle_array.weight);

    thrust::device_vector<float> joint_weight_array;
    joint_weight_array.insert(joint_weight_array.end(), persistent_weights, persistent_weights + particle_count);
    joint_weight_array.insert(joint_weight_array.end(), new_born_weights, new_born_weights + new_born_particle_count);

    thrust::device_vector<float> joint_weight_accum(joint_weight_array.size());
    accumulate(joint_weight_array, joint_weight_accum);

    float joint_max = joint_weight_accum.back();

    resamplingGenerateRandomNumbersKernel<<<particles_grid, block_dim>>>(rand_array, rng_states, joint_max,
                                                                         particle_count);

    CHECK_ERROR(hipGetLastError());
    // CHECK_ERROR(hipDeviceSynchronize());

    thrust::device_ptr<float> rand_ptr(rand_array);
    thrust::device_vector<float> rand_vector(rand_ptr, rand_ptr + particle_count);

    thrust::sort(rand_vector.begin(), rand_vector.end());

    thrust::device_vector<int> idx_resampled(particle_count);
    calc_resampled_indices(joint_weight_accum, rand_vector, idx_resampled, joint_max);
    int* idx_array_resampled = thrust::raw_pointer_cast(idx_resampled.data());

    float new_weight = joint_max / particle_count;

    resamplingKernel<<<particles_grid, block_dim>>>(particle_array, particle_array_next, birth_particle_array,
                                                    idx_array_resampled, new_weight, particle_count);

    CHECK_ERROR(hipGetLastError());
}

} /* namespace dogm */
